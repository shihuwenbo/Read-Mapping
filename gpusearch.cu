#include "hip/hip_runtime.h"

#include "misc.h"
#include "gpusearch.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

static hipError_t crc;
int nblock_size = 512;
int maxgsx = 65535;

__device__ char *kmbwt, *kmsr;
__device__ unsigned int *kmocc, *kmpsa;
__device__ unsigned int *kmsml;

// given a letter, return its rank in the alphabet
__device__ inline
unsigned int alpha_rank(char l)
{
    // rank alphabetically $<a<c<g<t
    switch(l)
    {
        case '$': return 0;
        case 'a': return 1;
        case 'c': return 2;
        case 'g': return 3;
        case 't': return 4;
    }
    return 5;
}

// get the bp at pos
__device__ inline
char gpu_get_bp_2bit(char* genome, unsigned int pos)
{
    unsigned long long int bit_pos = pos * ENCODE_SIZE_2BIT;
    unsigned long long int byte_pos = bit_pos / BYTE_SIZE;
    unsigned long long int byte_off = bit_pos - byte_pos * BYTE_SIZE;
    unsigned char dna = genome[byte_pos];
    dna = dna << byte_off;
    dna = dna >> (BYTE_SIZE - ENCODE_SIZE_2BIT);
    switch(dna)
    {   
        case 0: return 'a';
        case 1: return 'c';
        case 2: return 'g';
        case 3: return 't';
    }   
    return '*';
}

// write a bp at a position
__device__ inline
void gpu_write_bp_2bit(char* genome, unsigned int pos, char val)
{
    unsigned long long int bit_pos = pos * ENCODE_SIZE_2BIT;
    unsigned long long int byte_pos = bit_pos / BYTE_SIZE;
    unsigned long long int byte_off = bit_pos - byte_pos * BYTE_SIZE;
    switch(val)
    {   
        case 'a':
        {   
            char mask = 0 << (BYTE_SIZE - byte_off - ENCODE_SIZE_2BIT);
            genome[byte_pos] |= mask;
            break;
        }   
        case 'c':
        {   
            char mask = 1 << (BYTE_SIZE - byte_off - ENCODE_SIZE_2BIT);
            genome[byte_pos] |= mask;
            break;
        }   
        case 'g':
        {   
            char mask = 2 << (BYTE_SIZE - byte_off - ENCODE_SIZE_2BIT);
            genome[byte_pos] |= mask;
            break;
        }
        case 't':
        {
            char mask = 3 << (BYTE_SIZE - byte_off - ENCODE_SIZE_2BIT);
            genome[byte_pos] |= mask;
            break;
        }
    }

}

// given a bwt and a partial occ array, find the actual occ value
__device__ inline
unsigned int gpu_get_occ(long long int pos, char alpha)
{
    if(pos < 0)
        return 0;

    unsigned int occ_off = pos / SAMPLE_SIZE_TEST;

    long long int occi = 0;
    switch(alpha)
    {
        case 'a': occi = kmocc[occ_off*ALPHA_SIZE+0]; break;
        case 'c': occi = kmocc[occ_off*ALPHA_SIZE+1]; break;
        case 'g': occi = kmocc[occ_off*ALPHA_SIZE+2]; break;
        case 't': occi = kmocc[occ_off*ALPHA_SIZE+3]; break;
    }

    unsigned int ext_cnt = 0;
    for(unsigned int i = occ_off * SAMPLE_SIZE_TEST + 1;
        i < pos && i < (unsigned int)GENOME_SIZE_TEST; i++)
    {
        char bp = '*';
        if(i < BWT_DPOS)
            bp = gpu_get_bp_2bit(kmbwt, i);
        else if(i == BWT_DPOS)
            bp = '$';
        else
            bp = gpu_get_bp_2bit(kmbwt, i-1);
        if(bp == alpha)
            ext_cnt++;
    }

    occi += ext_cnt;

    if(pos % SAMPLE_SIZE_TEST == 0)
    {
        char bp = '*';
        if(pos < BWT_DPOS)
            bp = gpu_get_bp_2bit(kmbwt, pos);
        else if(pos == BWT_DPOS)
            bp = '$';
        else
            bp = gpu_get_bp_2bit(kmbwt, pos-1);
        if(bp == alpha)
            occi--;
        if(occi < 0)
            occi = 0;
    }
    return (unsigned int)occi;
}

// get suffix array value
__device__ inline
unsigned int gpu_get_sa_val(unsigned int pos)
{
    unsigned int nmov = 0;
    while(pos != BWT_DPOS)
    {
        if(pos % SAMPLE_SIZE_TEST == 0)
        {
            nmov += kmpsa[pos/SAMPLE_SIZE_TEST];
            break;
        }
        char bp = '*';
        if(pos < BWT_DPOS)
            bp = gpu_get_bp_2bit(kmbwt, pos);
        else if(pos == BWT_DPOS)
            bp = '$';
        else
            bp = gpu_get_bp_2bit(kmbwt, pos-1);
        pos = kmsml[alpha_rank(bp)] + 
            gpu_get_occ((long long int)pos,bp);
        nmov++;
    }
    return nmov;
}

// k-mismatch search
__device__ inline
void gpu_kmismatch(int kerr,unsigned int re,
              unsigned int sp,unsigned int ep,unsigned int* ans)
{
    /*
    for(long long int i = re; i >= (long long int)rs && ep >= sp; i--)
    {
        char bp = gpu_get_bp_2bit(sr, i);
        sp = sml[alpha_rank(bp)] + 1 +
             gpu_get_occ(bwt,occ,genome_size,sample_size,(long long int)sp-1,bp);
        ep = sml[alpha_rank(bp)] +
             gpu_get_occ(bwt,occ,genome_size,sample_size,(long long int)ep,bp);
    }
    if(sp <= ep)
        ans[0] = psa[sp-1];
    return 0;
    */
    if(sp > ep)
        return;

    if(re % READ_SIZE == 0)
    {
        ans[0] = gpu_get_sa_val(sp-1)-1;
        return;
    }

    unsigned int sp2 = kmsml[alpha_rank('a')] + 1 +
        gpu_get_occ((long long int)sp-1,'a');
    unsigned int ep2 = kmsml[alpha_rank('a')] +
        gpu_get_occ((long long int)ep,'a');
    int kerr2 = 0;
    if(gpu_get_bp_2bit(kmsr, re) != 'a')
        kerr2 = kerr - 1;
    else
        kerr2 = kerr;
    if(kerr2 >= 0)
        gpu_kmismatch(kerr2,re-1,sp2,ep2,ans);

    sp2 = kmsml[alpha_rank('c')] + 1 +
        gpu_get_occ((long long int)sp-1,'c');
    ep2 = kmsml[alpha_rank('c')] +
        gpu_get_occ((long long int)ep,'c');
    if(gpu_get_bp_2bit(kmsr, re) != 'c')
        kerr2 = kerr - 1;
    else
        kerr2 = kerr;
    if(kerr2 >= 0)
        gpu_kmismatch(kerr2,re-1,sp2,ep2,ans);

    sp2 = kmsml[alpha_rank('g')] + 1 +
        gpu_get_occ((long long int)sp-1,'g');
    ep2 = kmsml[alpha_rank('g')] +
        gpu_get_occ((long long int)ep,'g');
    if(gpu_get_bp_2bit(kmsr, re) != 'g')
        kerr2 = kerr - 1;
    else
        kerr2 = kerr;
    if(kerr2 >= 0)
        gpu_kmismatch(kerr2,re-1,sp2,ep2,ans);

    sp2 = kmsml[alpha_rank('t')] + 1 +
        gpu_get_occ((long long int)sp-1,'t');
    ep2 = kmsml[alpha_rank('t')] +
        gpu_get_occ((long long int)ep,'t');
    if(gpu_get_bp_2bit(kmsr, re) != 't')
        kerr2 = kerr - 1;
    else
        kerr2 = kerr;
    if(kerr2 >= 0)
        gpu_kmismatch(kerr2,re-1,sp2,ep2,ans);

    return;
}

// given a file of short reads, search it in genome, store result in all_ans
// maximum result is limited by ans_size
// maximum mismatch is kerr
__global__
void search_kernel(char* bwt, char* sr, unsigned int* psa,
        unsigned int read_num, unsigned int read_size, unsigned int* sml,
        unsigned int* occ, unsigned int* all_ans, int kerr)
{
    unsigned int i = (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(i < 100000)
    {
        kmbwt = bwt;
        kmocc = occ;
        kmpsa = psa;
        kmsr = sr; 
        kmsml = sml;
        unsigned int* ans_ptr = all_ans + i;
        unsigned int re = read_size*(i+1)-1;
        unsigned int sp = 0;
        unsigned int ep = (unsigned int)GENOME_SIZE_TEST - 1;
        gpu_kmismatch(kerr,re,sp,ep,ans_ptr);
    }
}

// c interface
extern "C++" void gpu_search(char* bwt, char* sr, unsigned int* psa,
        unsigned int read_num, unsigned int read_size, unsigned int* sml,
        unsigned int* occ, unsigned int* all_ans, int kerr)
{
    unsigned int x_size, y_size;
    dim3 dimBlock(nblock_size);
    x_size = (100000 - 1)/nblock_size + 1; 
    y_size = (x_size - 1)/maxgsx + 1;  
    x_size = x_size < maxgsx ? x_size : maxgsx;
    dim3 dimGrid(x_size, y_size);
    crc = hipGetLastError();
    search_kernel<<<dimGrid, dimBlock>>>
      (bwt,sr,psa,read_num,read_size,sml,occ,all_ans,kerr);
    hipDeviceSynchronize();
    crc = hipGetLastError();
    if (crc)
    {
        printf("gpusearch error=%d:%s\n",crc,hipGetErrorString(crc));
        exit(1);
    }   
}

// functions for memory copy allocation and deallocation
extern "C++" void gmalloc(char** gptr, size_t size)
{
    hipMalloc(gptr, size);
    return;
}

extern "C++" void gmemcpy_htod(char* dst, char* src, size_t size)
{
   hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
   return;
}

extern "C++" void gmemcpy_dtoh(char* dst, char* src, size_t size)
{
   hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
   return;
}

extern "C++" void gfree(char* src)
{
   hipFree(src);
   return;
}

extern "C++" void set_stack_size(size_t size)
{
   cudaThreadSetLimit(hipLimitStackSize, size); 
   return;
}

extern "C++" void set_cache_size(int nscache)
{
   hipFuncCache_t cpref;
   if ((nscache < 0) || (nscache > 2))
      return;
   if (nscache==0)
      cpref = hipFuncCachePreferNone;
   else if (nscache==1)
      cpref = hipFuncCachePreferShared;
   else if (nscache==2)
      cpref = hipFuncCachePreferL1;
   hipDeviceSetCacheConfig(cpref);
   return;
}
