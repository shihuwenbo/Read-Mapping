#include "hip/hip_runtime.h"
#include "misc.h"
#include "gpusearch.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

static hipError_t crc;

int nblock_size = 64;
int maxgsx = 65535;

// given a letter, return its rank in the alphabet
__device__
unsigned int alpha_rank(char l)
{
    // rank alphabetically $<a<c<g<t
    switch(l)
    {
        case '$': return 0;
        case 'a': return 1;
        case 'c': return 2;
        case 'g': return 3;
        case 't': return 4;
    }
    return 5;
}

// get the bp at pos
__device__
char gpu_get_bp_2bit(char* genome, unsigned int pos)
{
    unsigned long long int bit_pos = pos * ENCODE_SIZE_2BIT;
    unsigned long long int byte_pos = bit_pos / BYTE_SIZE;
    unsigned long long int byte_off = bit_pos - byte_pos * BYTE_SIZE;
    unsigned char dna = genome[byte_pos];
    dna = dna << byte_off;
    dna = dna >> (BYTE_SIZE - ENCODE_SIZE_2BIT);
    switch(dna)
    {   
        case 0: return 'a';
        case 1: return 'c';
        case 2: return 'g';
        case 3: return 't';
    }   
    return '*';
}

// given a bwt and a partial occ array, find the actual occ value
__device__
unsigned int gpu_get_occ(char* bwt, unsigned int* occ, unsigned int genome_size,
        unsigned int sample_size, long long int pos, char alpha)
{
    if(pos < 0)
        return 0;

    unsigned int occ_off = pos / sample_size;
    unsigned int* base_ptr = &occ[occ_off*ALPHA_SIZE];

    long long int occi = 0;
    switch(alpha)
    {
        case 'a': occi = base_ptr[0]; break;
        case 'c': occi = base_ptr[1]; break;
        case 'g': occi = base_ptr[2]; break;
        case 't': occi = base_ptr[3]; break;
    }

    unsigned int ext_cnt = 0;
    for(unsigned int i = occ_off * sample_size + 1;
        i < pos && i < genome_size; i++)
    {
        char bp = '*';
        if(i < BWT_DPOS)
            bp = gpu_get_bp_2bit(bwt, i);
        else if(i == BWT_DPOS)
            bp = '$';
        else
            bp = gpu_get_bp_2bit(bwt, i-1);
        if(bp == alpha)
            ext_cnt++;
    }

    occi += ext_cnt;

    if(pos % sample_size == 0)
    {
        char bp = '*';
        if(pos < BWT_DPOS)
            bp = gpu_get_bp_2bit(bwt, pos);
        else if(pos == BWT_DPOS)
            bp = '$';
        else
            bp = gpu_get_bp_2bit(bwt, pos-1);
        if(bp == alpha)
            occi--;
        if(occi < 0)
            occi = 0;
    }
    return (unsigned int)occi;
}

// get suffix array value
__device__
unsigned int gpu_get_sa_val(char* bwt, unsigned int*sml, unsigned int* occ,
        unsigned int*psa, unsigned int genome_size, unsigned int sample_size,
        unsigned int pos)
{
    unsigned int nmov = 0;
    int lucky = 0;
    while(pos != BWT_DPOS)
    {
        if(pos % sample_size == 0)
        {
            lucky = 1;
            break;
        }
        char bp = '*';
        if(pos < BWT_DPOS)
            bp = gpu_get_bp_2bit(bwt, pos);
        else if(pos == BWT_DPOS)
            bp = '$';
        else
            bp = gpu_get_bp_2bit(bwt, pos-1);
        pos = sml[alpha_rank(bp)] + 
            gpu_get_occ(bwt,occ,genome_size,sample_size,(long long int)pos,bp);
        nmov++;
    }
    if(lucky)
        nmov += psa[pos/sample_size];
    return nmov;
}

// k-mismatch search
__device__
unsigned int gpu_kmismatch(char* sr, char* bwt, unsigned int* sml,
        unsigned int* occ, unsigned int* psa, int kerr,
        unsigned int rs, unsigned int re, unsigned int sp, unsigned int ep,
        unsigned int* ans, unsigned int ans_cnt, unsigned int ans_size,
        unsigned int genome_size, unsigned int sample_size,
        const char *alphabet)
{
    /*
    for(long long int i = re; i >= (long long int)rs && ep >= sp; i--)
    {
        char bp = gpu_get_bp_2bit(sr, i);
        sp = sml[alpha_rank(bp)] + 1 +
             gpu_get_occ(bwt,occ,genome_size,sample_size,(long long int)sp-1,bp);
        ep = sml[alpha_rank(bp)] +
             gpu_get_occ(bwt,occ,genome_size,sample_size,(long long int)ep,bp);
    }
    if(sp <= ep)
        ans[0] = psa[sp-1];
    return 0;
    */
    if(sp > ep && ans_cnt < ans_size)
        return 0;
    if(re <= rs)
    {
        unsigned int cnt = 0;
        for(unsigned int i = sp - 1, j = ans_cnt; 
            i <= ep - 1 && j < ans_size; i++, j++)
        {
            ans[j] = gpu_get_sa_val(bwt,sml,occ,psa,genome_size,sample_size,i)-1;
            cnt++;
        }
        return cnt;
    }
    unsigned int new_ans_cnt = ans_cnt;
    for(unsigned int i = 0; i < ALPHA_SIZE; i++)
    {
        unsigned int sp2 = sml[alpha_rank(alphabet[i])] + 1 +
            gpu_get_occ(bwt,occ,genome_size,sample_size,
                    (long long int)sp-1,alphabet[i]);
        unsigned int ep2 = sml[alpha_rank(alphabet[i])] +
            gpu_get_occ(bwt,occ,genome_size,sample_size,
                    (long long int)ep,alphabet[i]);

        int kerr2 = 0;
        if(gpu_get_bp_2bit(sr, re) != alphabet[i])
            kerr2 = kerr - 1;
        else
            kerr2 = kerr;
        if(kerr2 >= 0)
        {
            new_ans_cnt += gpu_kmismatch(sr,bwt,sml,occ,psa,kerr2,rs,re-1,sp2,
                        ep2,ans,new_ans_cnt,ans_size,genome_size,
                        sample_size,alphabet);
        }
    }
    return new_ans_cnt;
}

// given a file of short reads, search it in genome, store result in all_ans
// maximum result is limited by ans_size
// maximum mismatch is kerr
__global__
void search_kernel(char* bwt, unsigned int genome_size, char* sr, unsigned int* psa,
        unsigned int read_num, unsigned int read_size, unsigned int* sml,
        unsigned int* occ, unsigned int spsize, unsigned int* all_ans,
        unsigned int ans_size, int kerr)
{
    const char alphabet[] = "acgt";
    unsigned int i = (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(i < read_num)
    {
        unsigned int* ans_ptr = all_ans + i * ans_size;
        unsigned int rs = i*read_size;
        unsigned int re = (i+1)*read_size-1;
        gpu_kmismatch(sr,bwt,sml,occ,psa,kerr,rs,re,0,genome_size-1,ans_ptr,
            0,ans_size,genome_size,spsize,alphabet);
    }
}

// c interface
extern "C++" void gpu_search(char* bwt, unsigned int genome_size, char* sr, unsigned int* psa,
        unsigned int read_num, unsigned int read_size, unsigned int* sml,
        unsigned int* occ, unsigned int spsize, unsigned int* all_ans,
        unsigned int ans_size, int kerr)
{
    unsigned int x_size, y_size;
    dim3 dimBlock(nblock_size);
    x_size = (read_num - 1)/nblock_size + 1; 
    y_size = (x_size - 1)/maxgsx + 1;  
    x_size = x_size < maxgsx ? x_size : maxgsx;
    dim3 dimGrid(x_size, y_size);
    crc = hipGetLastError();
    search_kernel<<<dimGrid, dimBlock>>>(bwt,genome_size,sr,psa,read_num,read_size,
         sml,occ,spsize,all_ans,ans_size,kerr);
    hipDeviceSynchronize();
    crc = hipGetLastError();
    if (crc)
    {
        printf("gpusearch error=%d:%s\n",crc,hipGetErrorString(crc));
        exit(1);
    }   
}

// functions for memory copy allocation and deallocation
extern "C++" void gmalloc(char** gptr, size_t size)
{
    hipMalloc(gptr, size);
    return;
}

extern "C++" void gmemcpy_htod(char* dst, char* src, size_t size)
{
   hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
   return;
}

extern "C++" void gmemcpy_dtoh(char* dst, char* src, size_t size)
{
   hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
   return;
}

extern "C++" void gfree(char* src)
{
   hipFree(src);
   return;
}
